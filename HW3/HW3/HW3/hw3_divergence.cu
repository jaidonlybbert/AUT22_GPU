#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "../Util/cuda_helpers.h"
#include <stdio.h>
#include <stdlib.h>

hipError_t executeCuda(int width);

/*
    3 KERNELS FOR TESTING EFFECTS OF DIVERGENCE
*/

__global__ void no_divergence(int* res, int width)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int temp = col;

    if (width % 2 == 0) {
        temp += row * 10;
    }

    __syncthreads();
    
    res[row * width + col] = temp;

    return;
}

__global__ void single_branch_divergence(int *res, int width)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int temp = col;

    if (temp % 2 == 0) {
        temp += row * 10;
    }
    else {
        temp += row * 1000;
    }

    __syncthreads();

    res[row * width + col] = temp;
    
    return;
}

__global__ void nested_branch_divergence(int* res, int width)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int temp = col;

    if (temp % 2 == 0) { // do for every other column
        temp += row * 10;
        if (temp % 3 == 0) { // do for every 6th column
            temp += row*1000;
        }
    }

    __syncthreads();

    res[row * width + col] = temp;

    return;
}

int main()
{
    int width = 36;

    // Run kernel
    checkCuda(executeCuda(width));

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    checkCuda(hipDeviceReset());

    return 0;
}

hipError_t executeCuda(int width)
{
    int *res = (int *)malloc(width*width*sizeof(int));
    int *dev_res = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    checkCuda(hipSetDevice(0));
    // Allocate GPU memory for variable
    checkCuda(hipMalloc((void**)&dev_res, width*width*sizeof(int)));
    // Copy result from host memory to GPU buffers.
    checkCuda(hipMemcpy(dev_res, res, width*width*sizeof(int), hipMemcpyHostToDevice));

    // Launch a kernel on the GPU with one thread for each element.
    dim3 DimGrid(23, 36, 1); dim3 DimBlock(16, 16, 1);
    no_divergence<<<DimGrid, DimBlock >>>(dev_res, width);
    // Check for any errors launching the kernel
    checkCuda(hipGetLastError());
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    checkCuda(hipDeviceSynchronize());

    single_branch_divergence << <DimGrid, DimBlock >> > (dev_res, width);
    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());

    nested_branch_divergence << <DimGrid, DimBlock >> > (dev_res, width);
    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());

    // Copy output vector from GPU buffer to host memory.
    checkCuda(hipMemcpy(res, dev_res, width*width*sizeof(int), hipMemcpyDeviceToHost));


Error:
    hipFree(dev_res);
    
    return cudaStatus;
}
