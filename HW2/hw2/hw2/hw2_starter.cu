#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_IMPLEMENTATION // this is needed
#include "../util/stb_image.h"  // download from class website files
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../util/stb_image_write.h"  // download from class website files

// #include your error-check macro header file here
#include "../util/cuda_helpers.h"

//#ifndef __HIPCC__  
//#define __HIPCC__
//#endif
//#include <hip/device_functions.h>

// global gaussian blur filter coefficients array here
#define BLUR_FILTER_WIDTH 9  // 9x9 (square) Gaussian blur filter
const float BLUR_FILT[81] = { 0.1084,0.1762,0.2494,0.3071,0.3292,0.3071,0.2494,0.1762,0.1084,0.1762,0.2865,0.4054,0.4994,0.5353,0.4994,0.4054,0.2865,0.1762,0.2494,0.4054,0.5738,0.7066,0.7575,0.7066,0.5738,0.4054,0.2494,0.3071,0.4994,0.7066,0.8703,0.9329,0.8703,0.7066,0.4994,0.3071,0.3292,0.5353,0.7575,0.9329,1.0000,0.9329,0.7575,0.5353,0.3292,0.3071,0.4994,0.7066,0.8703,0.9329,0.8703,0.7066,0.4994,0.3071,0.2494,0.4054,0.5738,0.7066,0.7575,0.7066,0.5738,0.4054,0.2494,0.1762,0.2865,0.4054,0.4994,0.5353,0.4994,0.4054,0.2865,0.1762,0.1084,0.1762,0.2494,0.3071,0.3292,0.3071,0.2494,0.1762,0.1084};

// DEFINE your CUDA blur kernel function(s) here
// blur kernel #1 - global memory only
__global__ void blurKernelGlobalMemory(unsigned char* imgData, unsigned char* imgOut, float* blurFilt, int imgWidth, int imgHeight)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int filtPadding = (BLUR_FILTER_WIDTH - 1) / 2;

    if (col < imgWidth && row < imgHeight) {
        float pixFloatVal = 0.0;
        float pixNormalizeFactor = 0.0;
        int pixVal = 0;
        int pixels = 0;

        // Get the weighted average of the surrounding pixels using the gaussian blur filter
        for (int blurRow = -filtPadding; blurRow < filtPadding + 1; ++blurRow) {
            for (int blurCol = -filtPadding; blurCol < filtPadding + 1; ++blurCol) {
                int curRow = row + blurRow;
                int curCol = col + blurCol;
                // Verify we have a valid image pixel
                if (curRow > -1 && curRow < imgHeight && curCol > -1 && curCol < imgWidth) {
                    pixFloatVal += (float)(imgData[curRow * imgWidth + curCol] * blurFilt[(blurRow+filtPadding) * BLUR_FILTER_WIDTH + blurCol+filtPadding]);
                    pixNormalizeFactor += blurFilt[(blurRow+filtPadding) * BLUR_FILTER_WIDTH + blurCol+filtPadding]; // Accumulate a factor to normalize by
                }
            }
        }
        // Write our new pixel value out
        imgOut[row * imgWidth + col] = (unsigned char)(int)(pixFloatVal / pixNormalizeFactor);
    }
}

// blur kernel #2 - device shared memory (static alloc)
__global__ void blurKernelStaticMemory(unsigned char* imgData, unsigned char* imgOut, float* blurFilt, int imgWidth, int imgHeight)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int filtPadding = (BLUR_FILTER_WIDTH - 1) / 2;

    // Copy filter coefficients from global -> shared memory using first 81 threads of the block
    __shared__ float ds_blurFilt[BLUR_FILTER_WIDTH][BLUR_FILTER_WIDTH];
    if (threadIdx.x < 9 && threadIdx.y < 9) {
        ds_blurFilt[threadIdx.y][threadIdx.x] = blurFilt[threadIdx.y * BLUR_FILTER_WIDTH + threadIdx.x];
    }
    __syncthreads();

    // Apply the filter to the image
    if (col < imgWidth && row < imgHeight) {
        float pixFloatVal = 0.0;
        float pixNormalizeFactor = 0.0;
        int pixVal = 0;
        int pixels = 0;

        // Get the weighted average of the surrounding pixels using the gaussian blur filter
        for (int blurRow = -filtPadding; blurRow < filtPadding + 1; ++blurRow) {
            for (int blurCol = -filtPadding; blurCol < filtPadding + 1; ++blurCol) {
                int curRow = row + blurRow;
                int curCol = col + blurCol;
                // Verify we have a valid image pixel
                if (curRow > -1 && curRow < imgHeight && curCol > -1 && curCol < imgWidth) {
                    pixFloatVal += (float)(imgData[curRow * imgWidth + curCol] * ds_blurFilt[blurRow+filtPadding][blurCol+filtPadding]);
                    pixNormalizeFactor += ds_blurFilt[blurRow+filtPadding][blurCol+filtPadding]; // Accumulate a factor to normalize by
                }
            }
        }
        // Write our new pixel value out
        imgOut[row * imgWidth + col] = (unsigned char)(int)(pixFloatVal / pixNormalizeFactor);
    }
}


// blur kernel #2 - device shared memory (dynamic alloc)
extern __shared__ float s_blurFilt[];

__global__ void blurKernelDynamicMemory(unsigned char* imgData, unsigned char* imgOut, float* blurFilt, int imgWidth, int imgHeight)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int filtPadding = (BLUR_FILTER_WIDTH - 1) / 2;

    // Copy filter coefficients from global -> shared memory using first 81 threads of the block
    if (threadIdx.x < 9 && threadIdx.y < 9) {
        s_blurFilt[threadIdx.y * BLUR_FILTER_WIDTH + threadIdx.x] = blurFilt[threadIdx.y * BLUR_FILTER_WIDTH + threadIdx.x];
    }
    __syncthreads();

    // Apply the filter to the image
    if (col < imgWidth && row < imgHeight) {
        float pixFloatVal = 0.0;
        float pixNormalizeFactor = 0.0;
        int pixVal = 0;
        int pixels = 0;

        // Get the weighted average of the surrounding pixels using the gaussian blur filter
        int curRow = 0;
        int curCol = 0;
        for (int blurRow = -filtPadding; blurRow < filtPadding + 1; ++blurRow) {
            for (int blurCol = -filtPadding; blurCol < filtPadding + 1; ++blurCol) {
                curRow = row + blurRow;
                curCol = col + blurCol;
                // Verify we have a valid image pixel
                if (curRow > -1 && curRow < imgHeight && curCol > -1 && curCol < imgWidth) {
                    pixFloatVal += (float)(imgData[curRow * imgWidth + curCol] * s_blurFilt[(blurRow+filtPadding) * BLUR_FILTER_WIDTH + blurCol+filtPadding]);
                    pixNormalizeFactor += s_blurFilt[(blurRow+filtPadding) * BLUR_FILTER_WIDTH + blurCol+filtPadding]; // Accumulate a factor to normalize by
                }
            }
        }
        // Write our new pixel value out
        imgOut[row * imgWidth + col] = (unsigned char)(int)(pixFloatVal / pixNormalizeFactor);
    }
}


// EXTRA CREDIT
// define host sequential blur-kernel routine


int main()
{
    // read input image from file - be aware of image pixel bit-depth and resolution (horiz x vertical)
    const char filename[] = "../util/hw2_testimage1.png";
    int x_cols = 0;
    int y_rows = 0;
    int n_pixdepth = 0;
    unsigned char* h_imgData = stbi_load(filename, &x_cols, &y_rows, &n_pixdepth, 1);
    int imgSize = x_cols * y_rows * (int)sizeof(unsigned char);
    int imgWidth = x_cols;
    int imgHeight = y_rows;

    // setup additional host variables, allocate host memory as needed
    hipError_t cudaStatus;
    unsigned char* h_imgOut = (unsigned char*)malloc(imgSize);

    // setup timers
    hipEvent_t timer1_start, timer1_end, timer2_start, timer2_end;
    float timer1_elapsed, timer2_elapsed;
    cudaStatus = hipEventCreate(&timer1_start);
    cudaStatus = hipEventCreate(&timer2_start);
    cudaStatus = hipEventCreate(&timer1_end);
    cudaStatus = hipEventCreate(&timer2_end);

    // START timer #1
    cudaStatus = hipEventRecord(timer1_start);

    // allocate device memory
    unsigned char* dev_imageData = 0;
    unsigned char* dev_imageOut = 0;
    float* dev_blurFilt = 0;
    
    cudaStatus = hipMalloc((void**)&dev_imageData, imgSize);
    cudaStatus = hipMalloc((void**)&dev_imageOut, imgSize);
    cudaStatus = hipMalloc((void**)&dev_blurFilt, 81 * sizeof(float));

    // copy host data to device
    cudaStatus = hipMemcpy(dev_imageData, h_imgData, imgSize, hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_imageOut, h_imgOut, imgSize, hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_blurFilt, &BLUR_FILT[0], 81 * sizeof(float), hipMemcpyHostToDevice);

    // START timer #2
    cudaStatus = hipEventRecord(timer2_start);

    // launch kernel --- use appropriate heuristics to determine #threads/block and #blocks/grid to ensure coverage of your 2D data range
    dim3 DimGrid(imgWidth / 16 + 1, imgHeight / 16 + 1, 1);
    dim3 DimBlock(16, 16, 1);
    
    blurKernelStaticMemory<<<DimGrid, DimBlock>>>(dev_imageData, dev_imageOut, dev_blurFilt, imgWidth, imgHeight);

    // Check for any errors launching the kernel
    cudaStatus = checkCuda(hipGetLastError());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // call hipDeviceSynchronize() to wait for the kernel to finish, and return
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        goto Error;
    }
    
    // STOP timer #2
    cudaStatus = hipEventRecord(timer2_end);
    // retrieve result data from device back to host
    cudaStatus = hipMemcpy(h_imgOut, dev_imageOut, imgSize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // STOP timer #1
    cudaStatus = hipEventRecord(timer1_end);

    // hipDeviceReset( ) must be called in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.

    // save result output image data to file

    const char imgFileOut[] = "../util/hw2_outimage1.png";
    stbi_write_png(imgFileOut, x_cols, y_rows, 1, h_imgOut, x_cols * n_pixdepth);


    // EXTRA CREDIT:
    // start timer #3
    // run host sequential blur routine
    // stop timer #3

    // retrieve and save timer results (write to console or file)
    cudaStatus = hipEventSynchronize(timer1_end);
    cudaStatus = hipEventSynchronize(timer2_end);
    cudaStatus = hipEventElapsedTime(&timer1_elapsed, timer1_start, timer1_end);
    cudaStatus = hipEventElapsedTime(&timer2_elapsed, timer2_start, timer2_end);
    cudaStatus = hipEventDestroy(timer1_start);
    cudaStatus = hipEventDestroy(timer2_start);
    cudaStatus = hipEventDestroy(timer1_end);
    cudaStatus = hipEventDestroy(timer2_end);
    fprintf(stderr, "Timer 1 elapsed: %2f\n", timer1_elapsed);
    fprintf(stderr, "Timer 2 elapsed: %2f\n", timer2_elapsed);
 
Error:  // assumes error macro has a goto Error statement

    // free host and device memory
    hipFree(dev_blurFilt);
    hipFree(dev_imageData);
    hipFree(dev_imageOut);

    return 0;
}


